#include "hip/hip_runtime.h"
#include"FD2DGPU.cuh"
__global__ void born_vx(float *txx,float *txz,float *vx,float *vx_x,float *vx_z,float *b_txx,float *b_txz,float *b_vx,float *b_vx_x,float *b_vx_z,\
                const int nxpml, const int nzpml,const float dt,const float dx,const float dz,\
				const int nop,float *rho,float *dampx,float *dampz,int direction)
{

	const int iz = blockIdx.x*blockDim.x+threadIdx.x;
	const int ix = blockIdx.y*blockDim.y+threadIdx.y;
	if(iz>nzpml-nop||ix>nxpml-nop||iz<nop||ix<nop)return;
    const int index = ix*nzpml+iz;    
//	__syncthreads();
		float damp1 = 1 - dt*dampx[index]/2;
		float damp2 = 1 + dt*dampx[index]/2;
		float damp3 = 1 - dt*dampz[index]/2;
		float damp4 = 1 + dt*dampz[index]/2;

		float tmp_txx = 0;
		float tmp_txz = 0;
#pragma unroll 4
		for(int i=1;i<=nop;i++)
		{
			tmp_txx += coeff2[i]*(txx[(ix+i-1)*nzpml+iz]-txx[(ix-i)*nzpml+iz]);
			tmp_txz += coeff2[i]*(txz[ix*nzpml+(iz+i-1)]-txz[ix*nzpml+(iz-i)]);
		}
		
		vx_x[index] = (damp1*vx_x[index]+direction*(1.0/rho[index])*(dt/dx)*tmp_txx)/damp2;
		vx_z[index] = (damp3*vx_z[index]+direction*(1.0/rho[index])*(dt/dz)*tmp_txz)/damp4;
		vx[index] = vx_x[index] + vx_z[index];

// born forward
		tmp_txx = 0;
		tmp_txz = 0;
#pragma unroll 4
		for(int i=1;i<=nop;i++)
		{
			tmp_txx += coeff2[i]*(b_txx[(ix+i-1)*nzpml+iz]-b_txx[(ix-i)*nzpml+iz]);
			tmp_txz += coeff2[i]*(b_txz[ix*nzpml+(iz+i-1)]-b_txz[ix*nzpml+(iz-i)]);
		}
		
		b_vx_x[index] = (damp1*b_vx_x[index]+direction*(1.0/rho[index])*(dt/dx)*tmp_txx)/damp2;
		b_vx_z[index] = (damp3*b_vx_z[index]+direction*(1.0/rho[index])*(dt/dz)*tmp_txz)/damp4;
		b_vx[index] = b_vx_x[index] + b_vx_z[index];

//
}



__global__ void born_vz(float *tzz,float *txz,float *vz,float *vz_x,float *vz_z,float *b_tzz,float *b_txz,float *b_vz,float *b_vz_x,float *b_vz_z,\
                const int nxpml, const int nzpml,const float dt,const float dx,const float dz,\
				const int nop,float *rho,float *dampx,float *dampz,int direction)
{

	const int iz = blockIdx.x*blockDim.x+threadIdx.x;
	const int ix = blockIdx.y*blockDim.y+threadIdx.y;
	if(iz>nzpml-nop||ix>nxpml-nop||iz<nop||ix<nop)return;
    const int index = ix*nzpml+iz;    
//	__syncthreads();
		float damp1 = 1 - dt*dampx[index]/2;
		float damp2 = 1 + dt*dampx[index]/2;
		float damp3 = 1 - dt*dampz[index]/2;
		float damp4 = 1 + dt*dampz[index]/2;

		float tmp_tzz = 0;
		float tmp_txz = 0;
#pragma unroll 4
		for(int i=1;i<=nop;i++)
		{
			tmp_tzz += coeff2[i]*(tzz[ix*nzpml+(iz+i)]-tzz[ix*nzpml+(iz-i+1)]);
			tmp_txz += coeff2[i]*(txz[(ix+i)*nzpml+iz]-txz[(ix-i+1)*nzpml+iz]);
		}
		
		vz_x[index] = (damp1*vz_x[index]+direction*(1.0/rho[index])*(dt/dx)*tmp_txz)/damp2;		
		vz_z[index] = (damp3*vz_z[index]+direction*(1.0/rho[index])*(dt/dz)*tmp_tzz)/damp4;
		vz[index] = vz_x[index] + vz_z[index];

//born forward
		tmp_tzz = 0;
		tmp_txz = 0;
#pragma unroll 4
		for(int i=1;i<=nop;i++)
		{
			tmp_tzz += coeff2[i]*(b_tzz[ix*nzpml+(iz+i)]-b_tzz[ix*nzpml+(iz-i+1)]);
			tmp_txz += coeff2[i]*(b_txz[(ix+i)*nzpml+iz]-b_txz[(ix-i+1)*nzpml+iz]);
		}
		
		b_vz_x[index] = (damp1*b_vz_x[index]+direction*(1.0/rho[index])*(dt/dx)*tmp_txz)/damp2;		
		b_vz_z[index] = (damp3*b_vz_z[index]+direction*(1.0/rho[index])*(dt/dz)*tmp_tzz)/damp4;
		b_vz[index] = b_vz_x[index] + b_vz_z[index];

}


__global__ void born_txx_tzz(float *txx, float *tzz,float *txx_x,float *txx_z,float *tzz_x,float *tzz_z,float *vx,float *vz,\
    float *b_txx, float *b_tzz,float *b_txx_x,float *b_txx_z,float *b_tzz_x,float *b_tzz_z,float *b_vx,float *b_vz,float *delta_mp,float *delta_ms,float *vp,float *vs,\
    const int nxpml, const int nzpml,const float dt,const float dx,const float dz,const int nop,float *lamda,float *miu,float* dampx,float* dampz,int direction)
{

	const int iz = blockIdx.x*blockDim.x+threadIdx.x;
	const int ix = blockIdx.y*blockDim.y+threadIdx.y;
	if(iz>nzpml-nop||ix>nxpml-nop||iz<nop||ix<nop)return;
    const int index = ix*nzpml+iz;
//	__syncthreads();

		float damp1 = 1 - dt*dampx[index]/2;
		float damp2 = 1 + dt*dampx[index]/2;
		float damp3 = 1 - dt*dampz[index]/2;
		float damp4 = 1 + dt*dampz[index]/2;

		float tmp_vx = 0;
		float tmp_vz = 0;

#pragma unroll 4
		for(int i=1;i<=nop;i++)
		{
			tmp_vx += coeff2[i]*(vx[(ix+i)*nzpml+iz]-vx[(ix-i+1)*nzpml+iz]);
			tmp_vz += coeff2[i]*(vz[ix*nzpml+(iz+i-1)]-vz[ix*nzpml+(iz-i)]);
		}
		 txx_x[index] = (damp1*txx_x[index]+direction*(lamda[index]+2*miu[index])*(dt/dx)*tmp_vx)/damp2;
		 txx_z[index] = (damp3*txx_z[index]+direction*lamda[index]*(dt/dz)*tmp_vz)/damp4;
		 tzz_x[index] = (damp1*tzz_x[index]+direction*lamda[index]*(dt/dx)*tmp_vx)/damp2;
		 tzz_z[index] = (damp3*tzz_z[index]+direction*(lamda[index]+2*miu[index])*(dt/dz)*tmp_vz)/damp4;
		 txx[index] = txx_x[index] + txx_z[index];
 		 tzz[index] = tzz_x[index] + tzz_z[index];

//add virtual source 
        float scale = 1.0;
        // float scale = 1.0*dt; 
		// b_txx_x[index] += scale*2*(lamda[index]+2*miu[index])*(delta_mp[index]/vp[index])*tmp_vx/dx;							//TODO: divide maybe not necessary
		// b_txx_z[index] += scale*2*((lamda[index]+2*miu[index])*(delta_mp[index]/vp[index])-2*miu[index]*delta_ms[index]/vs[index])*tmp_vz/dz;
		// b_tzz_x[index] += scale*2*((lamda[index]+2*miu[index])*(delta_mp[index]/vp[index])-2*miu[index]*delta_ms[index]/vs[index])*tmp_vx/dx;
		// b_tzz_z[index] += scale*2*(lamda[index]+2*miu[index])*(delta_mp[index]/vp[index])*tmp_vz/dz;

		b_txx_x[index] += scale*2*(lamda[index]+2*miu[index])*(delta_mp[index])*tmp_vx/dx;							//TODO: divide maybe not necessary
		b_txx_z[index] += scale*2*((lamda[index]+2*miu[index])*(delta_mp[index])-2*miu[index]*delta_ms[index])*tmp_vz/dz;
		b_tzz_x[index] += scale*2*((lamda[index]+2*miu[index])*(delta_mp[index])-2*miu[index]*delta_ms[index])*tmp_vx/dx;
		b_tzz_z[index] += scale*2*(lamda[index]+2*miu[index])*(delta_mp[index])*tmp_vz/dz;		
//
		tmp_vx = 0;
		tmp_vz = 0;

#pragma unroll 4
		for(int i=1;i<=nop;i++)
		{
			tmp_vx += coeff2[i]*(b_vx[(ix+i)*nzpml+iz]-b_vx[(ix-i+1)*nzpml+iz]);
			tmp_vz += coeff2[i]*(b_vz[ix*nzpml+(iz+i-1)]-b_vz[ix*nzpml+(iz-i)]);
		}
		 b_txx_x[index] = (damp1*b_txx_x[index]+direction*(lamda[index]+2*miu[index])*(dt/dx)*tmp_vx)/damp2;
		 b_txx_z[index] = (damp3*b_txx_z[index]+direction*lamda[index]*(dt/dz)*tmp_vz)/damp4;
		 b_tzz_x[index] = (damp1*b_tzz_x[index]+direction*lamda[index]*(dt/dx)*tmp_vx)/damp2;
		 b_tzz_z[index] = (damp3*b_tzz_z[index]+direction*(lamda[index]+2*miu[index])*(dt/dz)*tmp_vz)/damp4;
		 b_txx[index] = b_txx_x[index] + b_txx_z[index];
 		 b_tzz[index] = b_tzz_x[index] + b_tzz_z[index];


}


__global__ void born_txz(float *txz,float *txz_x,float *txz_z,float *vx,float *vz,float *b_txz,float *b_txz_x,float *b_txz_z,float *b_vx,float *b_vz,float *delta_ms, float *vs,\ 
    const int nxpml, const int nzpml,const float dt,const float dx,const float dz,const int nop,float *miu,const float* dampx,float *dampz,int direction)
{

	const int iz = blockIdx.x*blockDim.x+threadIdx.x;
	const int ix = blockIdx.y*blockDim.y+threadIdx.y;
	if(iz>nzpml-nop||ix>nxpml-nop||iz<nop||ix<nop)return;
    const int index = ix*nzpml+iz;
//	__syncthreads();

		float damp1 = 1 - dt*dampx[index]/2;
		float damp2 = 1 + dt*dampx[index]/2;
		float damp3 = 1 - dt*dampz[index]/2;
		float damp4 = 1 + dt*dampz[index]/2;

		float tmp_vx = 0;
		float tmp_vz = 0;

#pragma unroll 4
		for(int i=1;i<=nop;i++)
		{
			tmp_vx += coeff2[i]*(vx[ix*nzpml+(iz+i)]-vx[ix*nzpml+(iz-i+1)]);
			tmp_vz += coeff2[i]*(vz[(ix+i-1)*nzpml+iz]-vz[(ix-i)*nzpml+iz]);
		}
		
        txz_x[index] = (damp1*txz_x[index]+direction*(dt/dx)*miu[index]*tmp_vz)/damp2;
        txz_z[index] = (damp3*txz_z[index]+direction*(dt/dz)*miu[index]*tmp_vx)/damp4;
        txz[index] = txz_x[index] + txz_z[index];

//add  virtual source
        float scale = 1.0;
        // float scale = 1.0*dt;
		// b_txz_x[index] += scale*2*miu[index]*(delta_ms[index]/vs[index])*tmp_vz/dx;
		// b_txz_z[index] += scale*2*miu[index]*(delta_ms[index]/vs[index])*tmp_vx/dz;	

		b_txz_x[index] += scale*2*miu[index]*(delta_ms[index])*tmp_vz/dx;
		b_txz_z[index] += scale*2*miu[index]*(delta_ms[index])*tmp_vx/dz;	
//
		tmp_vx = 0;
		tmp_vz = 0;
#pragma unroll 4
		for(int i=1;i<=nop;i++)
		{
			tmp_vx += coeff2[i]*(b_vx[ix*nzpml+(iz+i)]-b_vx[ix*nzpml+(iz-i+1)]);
			tmp_vz += coeff2[i]*(b_vz[(ix+i-1)*nzpml+iz]-b_vz[(ix-i)*nzpml+iz]);
		}
		
		  b_txz_x[index] = (damp1*b_txz_x[index]+direction*(dt/dx)*miu[index]*tmp_vz)/damp2;
		  b_txz_z[index] = (damp3*b_txz_z[index]+direction*(dt/dz)*miu[index]*tmp_vx)/damp4;
		  b_txz[index] = b_txz_x[index] + b_txz_z[index];

}

//////////////////////////////////////////////////////////////////////////////////////////////////////
// rewrite born modeling schedue //////
/////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void update_vx(float *txx,float *txz,float *vx,float *vx_x,float *vx_z,\
                const int nxpml, const int nzpml,const float dt,const float dx,const float dz,\
				const int nop,float *rho,float *dampx,float *dampz,int direction)
{

	const int iz = blockIdx.x*blockDim.x+threadIdx.x;
	const int ix = blockIdx.y*blockDim.y+threadIdx.y;
	if(iz>=nzpml-nop||ix>=nxpml-nop||iz<nop||ix<nop)return;
    const int index = ix*nzpml+iz;    
//	__syncthreads();
		float damp1 = 1 - dt*dampx[index]/2;
		float damp2 = 1 + dt*dampx[index]/2;
		float damp3 = 1 - dt*dampz[index]/2;
		float damp4 = 1 + dt*dampz[index]/2;

		float tmp_txx = 0;
		float tmp_txz = 0;
#pragma unroll 4
		for(int i=1;i<=nop;i++)
		{
//			tmp_txx += coeff2[i]*(txx[(ix+i-1)*nzpml+iz]-txx[(ix-i)*nzpml+iz]);
//			tmp_txz += coeff2[i]*(txz[ix*nzpml+(iz+i-1)]-txz[ix*nzpml+(iz-i)]);
			 tmp_txx += coeff2[i]*(txx[(ix+i)*nzpml+iz]-txx[(ix-i+1)*nzpml+iz]);
			 tmp_txz += coeff2[i]*(txz[ix*nzpml+(iz+i-1)]-txz[ix*nzpml+(iz-i)]);			
		}
		
		vx_x[index] = (damp1*vx_x[index]+direction*(1.0/rho[index])*(dt/dx)*tmp_txx)/damp2;
		vx_z[index] = (damp3*vx_z[index]+direction*(1.0/rho[index])*(dt/dz)*tmp_txz)/damp4;
		vx[index] = vx_x[index] + vx_z[index];

}



__global__ void update_vz(float *tzz,float *txz,float *vz,float *vz_x,float *vz_z,\
                const int nxpml, const int nzpml,const float dt,const float dx,const float dz,\
				const int nop,float *rho,float *dampx,float *dampz,int direction)
{

	const int iz = blockIdx.x*blockDim.x+threadIdx.x;
	const int ix = blockIdx.y*blockDim.y+threadIdx.y;
	if(iz>=nzpml-nop||ix>=nxpml-nop||iz<nop||ix<nop)return;
    const int index = ix*nzpml+iz;    
//	__syncthreads();
		float damp1 = 1 - dt*dampx[index]/2;
		float damp2 = 1 + dt*dampx[index]/2;
		float damp3 = 1 - dt*dampz[index]/2;
		float damp4 = 1 + dt*dampz[index]/2;

		float tmp_tzz = 0;
		float tmp_txz = 0;
#pragma unroll 4
		for(int i=1;i<=nop;i++)
		{
//			tmp_tzz += coeff2[i]*(tzz[ix*nzpml+(iz+i)]-tzz[ix*nzpml+(iz-i+1)]);
//			tmp_txz += coeff2[i]*(txz[(ix+i)*nzpml+iz]-txz[(ix-i+1)*nzpml+iz]);
			 tmp_tzz += coeff2[i]*(tzz[ix*nzpml+(iz+i)]-tzz[ix*nzpml+(iz-i+1)]);
			 tmp_txz += coeff2[i]*(txz[(ix+i-1)*nzpml+iz]-txz[(ix-i)*nzpml+iz]);			
		}
		
		vz_x[index] = (damp1*vz_x[index]+direction*(1.0/rho[index])*(dt/dx)*tmp_txz)/damp2;		
		vz_z[index] = (damp3*vz_z[index]+direction*(1.0/rho[index])*(dt/dz)*tmp_tzz)/damp4;
		vz[index] = vz_x[index] + vz_z[index];

}


__global__ void update_txx_tzz(float *txx, float *tzz,float *txx_x,float *txx_z,float *tzz_x,float *tzz_z,float *vx,float *vz,\
    const int nxpml, const int nzpml,const float dt,const float dx,const float dz,const int nop,float *lamda,float *miu,float* dampx,float* dampz,int direction)
{

	const int iz = blockIdx.x*blockDim.x+threadIdx.x;
	const int ix = blockIdx.y*blockDim.y+threadIdx.y;
	if(iz>=nzpml-nop||ix>=nxpml-nop||iz<nop||ix<nop)return;
    const int index = ix*nzpml+iz;
//	__syncthreads();

		float damp1 = 1 - dt*dampx[index]/2;
		float damp2 = 1 + dt*dampx[index]/2;
		float damp3 = 1 - dt*dampz[index]/2;
		float damp4 = 1 + dt*dampz[index]/2;

		float tmp_vx = 0;
		float tmp_vz = 0;

#pragma unroll 4
		for(int i=1;i<=nop;i++)
		{
//			tmp_vx += coeff2[i]*(vx[(ix+i)*nzpml+iz]-vx[(ix-i+1)*nzpml+iz]);
//			tmp_vz += coeff2[i]*(vz[ix*nzpml+(iz+i-1)]-vz[ix*nzpml+(iz-i)]);
			 tmp_vx += coeff2[i]*(vx[(ix+i-1)*nzpml+iz]-vx[(ix-i)*nzpml+iz]);
			 tmp_vz += coeff2[i]*(vz[ix*nzpml+(iz+i-1)]-vz[ix*nzpml+(iz-i)]);			
		}
		 txx_x[index] = (damp1*txx_x[index]+direction*(lamda[index]+2*miu[index])*(dt/dx)*tmp_vx)/damp2;
		 txx_z[index] = (damp3*txx_z[index]+direction*lamda[index]*(dt/dz)*tmp_vz)/damp4;
		 tzz_x[index] = (damp1*tzz_x[index]+direction*lamda[index]*(dt/dx)*tmp_vx)/damp2;
		 tzz_z[index] = (damp3*tzz_z[index]+direction*(lamda[index]+2*miu[index])*(dt/dz)*tmp_vz)/damp4;
		 txx[index] = txx_x[index] + txx_z[index];
 		 tzz[index] = tzz_x[index] + tzz_z[index];

}


__global__ void update_txz(float *txz,float *txz_x,float *txz_z,float *vx,float *vz,const int nxpml, const int nzpml,\
	const float dt,const float dx,const float dz,const int nop,float *miu,float* dampx,float *dampz,int direction)
{

	const int iz = blockIdx.x*blockDim.x+threadIdx.x;
	const int ix = blockIdx.y*blockDim.y+threadIdx.y;
	if(iz>=nzpml-nop||ix>=nxpml-nop||iz<nop||ix<nop)return;
    const int index = ix*nzpml+iz;
//	__syncthreads();

		float damp1 = 1 - dt*dampx[index]/2;
		float damp2 = 1 + dt*dampx[index]/2;
		float damp3 = 1 - dt*dampz[index]/2;
		float damp4 = 1 + dt*dampz[index]/2;

		float tmp_vx = 0;
		float tmp_vz = 0;

#pragma unroll 4
		for(int i=1;i<=nop;i++)
		{
//			tmp_vx += coeff2[i]*(vx[ix*nzpml+(iz+i)]-vx[ix*nzpml+(iz-i+1)]);
//			tmp_vz += coeff2[i]*(vz[(ix+i-1)*nzpml+iz]-vz[(ix-i)*nzpml+iz]);
			 tmp_vx += coeff2[i]*(vx[ix*nzpml+(iz+i)]-vx[ix*nzpml+(iz-i+1)]);
			 tmp_vz += coeff2[i]*(vz[(ix+i)*nzpml+iz]-vz[(ix-i+1)*nzpml+iz]);			
		}
		
        txz_x[index] = (damp1*txz_x[index]+direction*(dt/dx)*miu[index]*tmp_vz)/damp2;
        txz_z[index] = (damp3*txz_z[index]+direction*(dt/dz)*miu[index]*tmp_vx)/damp4;
        txz[index] = txz_x[index] + txz_z[index];

}



// optimization finite-difference kernel 

__global__ void update_vx_vz(float *txx,float *tzz,float *txz,float *vx,float *vx_x,float *vx_z,float *vz,float *vz_x,float *vz_z,\
                const int nxpml, const int nzpml,const float dt,const float dx,const float dz,\
				const int nop,float *rho,float *dampx,float *dampz,int direction)
{

	const int iz = blockIdx.x*blockDim.x+threadIdx.x;
	const int ix = blockIdx.y*blockDim.y+threadIdx.y;
	if(iz>=nzpml-nop||ix>=nxpml-nop||iz<nop||ix<nop)return;
    const int index = ix*nzpml+iz;    
//	__syncthreads();
		float damp1 = 1 - dt*dampx[index]/2;
		float damp2 = 1 + dt*dampx[index]/2;
		float damp3 = 1 - dt*dampz[index]/2;
		float damp4 = 1 + dt*dampz[index]/2;

		float tmp_txx_x = 0;
		float tmp_txz_z = 0;

		float tmp_tzz_z = 0;
		float tmp_txz_x = 0;		
#pragma unroll 4
		for(int i=1;i<=nop;i++)
		{
			 tmp_txx_x += coeff2[i]*(txx[(ix+i)*nzpml+iz]-txx[(ix-i+1)*nzpml+iz]);
			 tmp_txz_z += coeff2[i]*(txz[ix*nzpml+(iz+i-1)]-txz[ix*nzpml+(iz-i)]);	

			 tmp_tzz_z += coeff2[i]*(tzz[ix*nzpml+(iz+i)]-tzz[ix*nzpml+(iz-i+1)]);
			 tmp_txz_x += coeff2[i]*(txz[(ix+i-1)*nzpml+iz]-txz[(ix-i)*nzpml+iz]);				 		
		}
		
		vx_x[index] = (damp1*vx_x[index]+direction*(1.0/rho[index])*(dt/dx)*tmp_txx_x)/damp2;
		vx_z[index] = (damp3*vx_z[index]+direction*(1.0/rho[index])*(dt/dz)*tmp_txz_z)/damp4;

		vz_x[index] = (damp1*vz_x[index]+direction*(1.0/rho[index])*(dt/dx)*tmp_txz_x)/damp2;		
		vz_z[index] = (damp3*vz_z[index]+direction*(1.0/rho[index])*(dt/dz)*tmp_tzz_z)/damp4;
	
		vx[index] = vx_x[index] + vx_z[index];
		vz[index] = vz_x[index] + vz_z[index];			

}




__global__ void update_txx_tzz_txz(float *txx, float *tzz,float *txx_x,float *txx_z,float *tzz_x,float *tzz_z,float *txz,float *txz_x,float *txz_z,float *vx,float *vz,\
    const int nxpml, const int nzpml,const float dt,const float dx,const float dz,const int nop,float *lamda,float *miu,float* dampx,float* dampz,int direction)
{

	const int iz = blockIdx.x*blockDim.x+threadIdx.x;
	const int ix = blockIdx.y*blockDim.y+threadIdx.y;
	if(iz>=nzpml-nop||ix>=nxpml-nop||iz<nop||ix<nop)return;
    const int index = ix*nzpml+iz;
//	__syncthreads();

		float damp1 = 1 - dt*dampx[index]/2;
		float damp2 = 1 + dt*dampx[index]/2;
		float damp3 = 1 - dt*dampz[index]/2;
		float damp4 = 1 + dt*dampz[index]/2;

		float tmp_vx_x = 0;
		float tmp_vz_z = 0;
		float tmp_vx_z = 0;
		float tmp_vz_x = 0;


#pragma unroll 4
		for(int i=1;i<=nop;i++)
		{
			 tmp_vx_x += coeff2[i]*(vx[(ix+i-1)*nzpml+iz]-vx[(ix-i)*nzpml+iz]);
			 tmp_vz_z += coeff2[i]*(vz[ix*nzpml+(iz+i-1)]-vz[ix*nzpml+(iz-i)]);	

			 tmp_vx_z += coeff2[i]*(vx[ix*nzpml+(iz+i)]-vx[ix*nzpml+(iz-i+1)]);
			 tmp_vz_x += coeff2[i]*(vz[(ix+i)*nzpml+iz]-vz[(ix-i+1)*nzpml+iz]);	

		}
		 txx_x[index] = (damp1*txx_x[index]+direction*(lamda[index]+2*miu[index])*(dt/dx)*tmp_vx_x)/damp2;
		 txx_z[index] = (damp3*txx_z[index]+direction*lamda[index]*(dt/dz)*tmp_vz_z)/damp4;
		 tzz_x[index] = (damp1*tzz_x[index]+direction*lamda[index]*(dt/dx)*tmp_vx_x)/damp2;
		 tzz_z[index] = (damp3*tzz_z[index]+direction*(lamda[index]+2*miu[index])*(dt/dz)*tmp_vz_z)/damp4;
         txz_x[index] = (damp1*txz_x[index]+direction*(dt/dx)*miu[index]*tmp_vz_x)/damp2;
         txz_z[index] = (damp3*txz_z[index]+direction*(dt/dz)*miu[index]*tmp_vx_z)/damp4;
	 
		 txx[index] = txx_x[index] + txx_z[index];
 		 tzz[index] = tzz_x[index] + tzz_z[index];
         txz[index] = txz_x[index] + txz_z[index];	
}









//reconstruct wavefield p
__global__ void update_txx_tzz_txz_grad(float *txx, float *tzz,float *txx_x,float *txx_z,float *tzz_x,float *tzz_z,float *txz,float *txz_x,float *txz_z,float *vx,float *vz,\
    float *vx_gx,float *vz_gz,float *vx_gz,float *vz_gx,const int nxpml, const int nzpml,const float dt,const float dx,const float dz,const int nop,float *lamda,float *miu,float* dampx,float* dampz,int direction,int pml)
{

	const int iz = blockIdx.x*blockDim.x+threadIdx.x;
	const int ix = blockIdx.y*blockDim.y+threadIdx.y;
	if(iz>=nzpml-nop||ix>=nxpml-nop||iz<nop||ix<nop)return;
    const int index = ix*nzpml+iz;
//	__syncthreads();

		float damp1 = 1 - dt*dampx[index]/2;
		float damp2 = 1 + dt*dampx[index]/2;
		float damp3 = 1 - dt*dampz[index]/2;
		float damp4 = 1 + dt*dampz[index]/2;

		float tmp_vx_x = 0;
		float tmp_vz_z = 0;
		float tmp_vx_z = 0;
		float tmp_vz_x = 0;


#pragma unroll 4
		for(int i=1;i<=nop;i++)
		{
			 tmp_vx_x += coeff2[i]*(vx[(ix+i-1)*nzpml+iz]-vx[(ix-i)*nzpml+iz]);
			 tmp_vz_z += coeff2[i]*(vz[ix*nzpml+(iz+i-1)]-vz[ix*nzpml+(iz-i)]);	

			 tmp_vx_z += coeff2[i]*(vx[ix*nzpml+(iz+i)]-vx[ix*nzpml+(iz-i+1)]);
			 tmp_vz_x += coeff2[i]*(vz[(ix+i)*nzpml+iz]-vz[(ix-i+1)*nzpml+iz]);	

		}
		 txx_x[index] = (damp1*txx_x[index]+direction*(lamda[index]+2*miu[index])*(dt/dx)*tmp_vx_x)/damp2;
		 txx_z[index] = (damp3*txx_z[index]+direction*lamda[index]*(dt/dz)*tmp_vz_z)/damp4;
		 tzz_x[index] = (damp1*tzz_x[index]+direction*lamda[index]*(dt/dx)*tmp_vx_x)/damp2;
		 tzz_z[index] = (damp3*tzz_z[index]+direction*(lamda[index]+2*miu[index])*(dt/dz)*tmp_vz_z)/damp4;
         txz_x[index] = (damp1*txz_x[index]+direction*(dt/dx)*miu[index]*tmp_vz_x)/damp2;
         txz_z[index] = (damp3*txz_z[index]+direction*(dt/dz)*miu[index]*tmp_vx_z)/damp4;
	 
		 txx[index] = txx_x[index] + txx_z[index];
 		 tzz[index] = tzz_x[index] + tzz_z[index];
         txz[index] = txz_x[index] + txz_z[index];	

		 vx_gx[iz+ix*nzpml] = tmp_vx_x/dx;
		 vz_gz[iz+ix*nzpml] = tmp_vz_z/dz;

		 vx_gz[iz+ix*nzpml] = tmp_vx_z/dz;
		 vz_gx[iz+ix*nzpml] = tmp_vz_x/dx;

}



// adjoint wavefield
__global__ void GPUcalculate_elastic_vx_vz_back(float *txx,float *tzz,float *txz,float *vx,float *vx_x,float *vx_z,float *vz,float *vz_x,float *vz_z,float *lamda,float *miu,const int nxpml, const int nzpml,const float dt,const float dx,const float dz,\
									const int nop,float *rho,float *dampx,float *dampz,int direction)
{

	const int iz = blockIdx.x*blockDim.x+threadIdx.x;
	const int ix = blockIdx.y*blockDim.y+threadIdx.y;
	if(iz>=nzpml-nop||ix>=nxpml-nop||iz<nop||ix<nop)return;
//	__syncthreads();
		float damp1 = 1 - dt*dampx[iz+ix*nzpml]/2;
		float damp2 = 1 + dt*dampx[iz+ix*nzpml]/2;
		float damp3 = 1 - dt*dampz[iz+ix*nzpml]/2;
		float damp4 = 1 + dt*dampz[iz+ix*nzpml]/2;

		float tmp_txx_x = 0;
		float tmp_txz_z = 0;
		float tmp_tzz_x = 0;

		float tmp_tzz_z = 0;
		float tmp_txz_x = 0;
		float tmp_txx_z = 0;	

///////////////////////// lamda and miu is out	of partial differential 	
// #pragma unroll 4
// 		for(int i=1;i<=nop;i++)
// 		{		
// 			tmp_txx += coeff2[i]*(txx[(ix+i)*nzpml+iz]-txx[(ix-i+1)*nzpml+iz]);
// 			tmp_txz += coeff2[i]*(txz[ix*nzpml+(iz+i-1)]-txz[ix*nzpml+(iz-i)]);
// 			tmp_tzz += coeff2[i]*(tzz[(ix+i)*nzpml+iz]-tzz[(ix-i+1)*nzpml+iz]);			
// 		}
		
// 		vx_x[iz+ix*nzpml] = (damp1*vx_x[iz+ix*nzpml]+direction*(1.0/rho[iz+ix*nzpml])*(lamda[iz+ix*nzpml]+2*miu[iz+ix*nzpml])*(dt/dx)*tmp_txx \
// 							+direction*(1.0/rho[iz+ix*nzpml])*(lamda[iz+ix*nzpml])*(dt/dx)*tmp_tzz)/damp2;
// 		vx_z[iz+ix*nzpml] = (damp3*vx_z[iz+ix*nzpml]+direction*(1.0/rho[iz+ix*nzpml])*miu[iz+ix*nzpml]*(dt/dz)*tmp_txz)/damp4;					
// 		vx[iz+ix*nzpml] = vx_x[iz+ix*nzpml] + vx_z[iz+ix*nzpml];				//TODO : add source   divide  rho

///////////////////////// lamda miu is in partial differential 
#pragma unroll 4
		for(int i=1;i<=nop;i++)
		{		
			tmp_txx_x += coeff2[i]*((lamda[(ix+i)*nzpml+iz]+2*miu[(ix+i)*nzpml+iz])*txx[(ix+i)*nzpml+iz]-(lamda[(ix-i+1)*nzpml+iz]+2*miu[(ix-i+1)*nzpml+iz])*txx[(ix-i+1)*nzpml+iz]);
			tmp_txz_z += coeff2[i]*(miu[ix*nzpml+(iz+i-1)]*txz[ix*nzpml+(iz+i-1)]-miu[ix*nzpml+(iz-i)]*txz[ix*nzpml+(iz-i)]);				
			tmp_tzz_x += coeff2[i]*((lamda[(ix+i)*nzpml+iz])*tzz[(ix+i)*nzpml+iz]-(lamda[(ix-i+1)*nzpml+iz])*tzz[(ix-i+1)*nzpml+iz]);			
			
			tmp_tzz_z += coeff2[i]*((lamda[ix*nzpml+(iz+i)]+2*miu[ix*nzpml+(iz+i)])*tzz[ix*nzpml+(iz+i)]-(lamda[ix*nzpml+(iz-i+1)]+2*miu[ix*nzpml+(iz-i+1)])*tzz[ix*nzpml+(iz-i+1)]);
			tmp_txz_x += coeff2[i]*(miu[(ix+i-1)*nzpml+iz]*txz[(ix+i-1)*nzpml+iz]-miu[(ix-i)*nzpml+iz]*txz[(ix-i)*nzpml+iz]);				//TODO: miu and txz are not in a same grid point
			tmp_txx_z += coeff2[i]*((lamda[ix*nzpml+(iz+i)])*txx[ix*nzpml+(iz+i)]-(lamda[ix*nzpml+(iz-i+1)])*txx[ix*nzpml+(iz-i+1)]);					
		}
		
		vx_x[iz+ix*nzpml] = (damp1*vx_x[iz+ix*nzpml]+direction*(1.0/rho[iz+ix*nzpml])*(dt/dx)*tmp_txx_x \
							+direction*(1.0/rho[iz+ix*nzpml])*(dt/dx)*tmp_tzz_x)/damp2;
		vx_z[iz+ix*nzpml] = (damp3*vx_z[iz+ix*nzpml]+direction*(1.0/rho[iz+ix*nzpml])*(dt/dz)*tmp_txz_z)/damp4;					
		
		vz_x[iz+ix*nzpml] = (damp1*vz_x[iz+ix*nzpml]+direction*(1.0/rho[iz+ix*nzpml])*(dt/dx)*tmp_txz_x)/damp2;		
		vz_z[iz+ix*nzpml] = (damp3*vz_z[iz+ix*nzpml]+direction*(1.0/rho[iz+ix*nzpml])*(dt/dz)*tmp_tzz_z \
							+direction*(1.0/rho[iz+ix*nzpml])*(dt/dz)*tmp_txx_z)/damp4;
		
		vz[iz+ix*nzpml] = vz_x[iz+ix*nzpml] + vz_z[iz+ix*nzpml];		
		vx[iz+ix*nzpml] = vx_x[iz+ix*nzpml] + vx_z[iz+ix*nzpml];	
}


__global__ void GPUcalculate_elastic_txx_tzz_txz_back(float *txx, float *tzz,float *txx_x,float *txx_z,float *tzz_x,float *tzz_z,float *txz,float *txz_x,float *txz_z,float *vx,float *vz,const int nxpml, const int nzpml,const float dt,const float dx,const float dz,\
	const int nop,float *lamda,float *miu,float* dampx,float* dampz,int direction)
{
	const int iz = blockIdx.x*blockDim.x+threadIdx.x;
	const int ix = blockIdx.y*blockDim.y+threadIdx.y;
	if(iz>=nzpml-nop||ix>=nxpml-nop||iz<nop||ix<nop)return;
//	__syncthreads();

		float damp1 = 1 - dt*dampx[iz+ix*nzpml]/2;
		float damp2 = 1 + dt*dampx[iz+ix*nzpml]/2;
		float damp3 = 1 - dt*dampz[iz+ix*nzpml]/2;
		float damp4 = 1 + dt*dampz[iz+ix*nzpml]/2;

		float tmp_vx_x = 0;
		float tmp_vz_z = 0;

		float tmp_vx_z = 0;
		float tmp_vz_x = 0;

#pragma unroll 4
		for(int i=1;i<=nop;i++)
		{
			tmp_vx_x += coeff2[i]*(vx[(ix+i-1)*nzpml+iz]-vx[(ix-i)*nzpml+iz]);
			tmp_vz_z += coeff2[i]*(vz[ix*nzpml+(iz+i-1)]-vz[ix*nzpml+(iz-i)]);

			tmp_vx_z += coeff2[i]*(vx[ix*nzpml+(iz+i)]-vx[ix*nzpml+(iz-i+1)]);
			tmp_vz_x += coeff2[i]*(vz[(ix+i)*nzpml+iz]-vz[(ix-i+1)*nzpml+iz]);			
		}
		
		 txx_x[iz+ix*nzpml] = (damp1*txx_x[iz+ix*nzpml]+direction*(dt/dx)*tmp_vx_x)/damp2;
		 tzz_z[iz+ix*nzpml] = (damp3*tzz_z[iz+ix*nzpml]+direction*(dt/dz)*tmp_vz_z)/damp4;
		 txz_x[iz+ix*nzpml] = (damp1*txz_x[iz+ix*nzpml]+direction*(dt/dx)*tmp_vz_x)/damp2;
		 txz_z[iz+ix*nzpml] = (damp3*txz_z[iz+ix*nzpml]+direction*(dt/dz)*tmp_vx_z)/damp4;

		 txx[iz+ix*nzpml] = txx_x[iz+ix*nzpml];
 		 tzz[iz+ix*nzpml] = tzz_z[iz+ix*nzpml];
		 txz[iz+ix*nzpml] = txz_x[iz+ix*nzpml] + txz_z[iz+ix*nzpml];	

}


// __global__ void add_born_source(float *vx,float *vz,float *b_txx, float *b_tzz,float *b_txz,float *delta_mp,float *delta_ms,\
//     const int nxpml, const int nzpml,const int pml,const float dx,const float dz,const int nop,float *lamda,float *miu,float *rho,float *vp,float *vs,int direction)
// {

// 	const int iz = blockIdx.x*blockDim.x+threadIdx.x;
// 	const int ix = blockIdx.y*blockDim.y+threadIdx.y;
// 	// if(iz>=nzpml-nop||ix>=nxpml-nop||iz<nop||ix<nop)return;
// 	if(iz>=nzpml-pml||ix>=nxpml-pml||iz<pml||ix<pml)return;
//     const int index = ix*nzpml+iz;
// //add virtual source 
// 		float scale =1.0;

// 		// b_txx[index] += scale*2*(lamda[index]+2*miu[index])*(delta_mp[index])*(vx[(ix)*nzpml+iz]-vx[(ix-1)*nzpml+iz])/dx + \
// 		// 	2*((lamda[index]+2*miu[index])*(delta_mp[index])-2*miu[index]*delta_ms[index])*(vz[ix*nzpml+(iz)]-vz[ix*nzpml+(iz-1)])/dz;

// 		// b_tzz[index] += scale*2*((lamda[index]+2*miu[index])*(delta_mp[index])-2*miu[index]*delta_ms[index])*(vx[(ix)*nzpml+iz]-vx[(ix-1)*nzpml+iz])/dx + \
// 		// 	2*(lamda[index]+2*miu[index])*(delta_mp[index])*(vz[ix*nzpml+(iz)]-vz[ix*nzpml+(iz-1)])/dz;

// 		// b_txz[index] += scale*2*miu[index]*(delta_ms[index])*((vz[(ix+1)*nzpml+iz]-vz[ix*nzpml+iz])/dx + (vx[ix*nzpml+(iz+1)]-vx[ix*nzpml+iz])/dz);

// ////////////////////
// 		// float tmp_vx_x = 0;
// 		// float tmp_vz_z = 0;
// 		// float tmp_vx_z = 0;
// 		// float tmp_vz_x = 0;

// 		// tmp_vx_x = (vx[(ix)*nzpml+iz]-vx[(ix-1)*nzpml+iz]);
// 		// tmp_vz_z = (vz[ix*nzpml+(iz)]-vz[ix*nzpml+(iz-1)]);			

// 		// tmp_vx_z = (vx[ix*nzpml+(iz+1)] + vx[(ix-1)*nzpml+(iz+1)]) - (vx[ix*nzpml+(iz-1)] + vx[(ix-1)*nzpml+(iz-1)]);
// 		// tmp_vz_x = (vz[(ix+1)*nzpml+iz] + vz[(ix+1)*nzpml+(iz-1)]) - (vz[(ix-1)*nzpml+iz] + vz[(ix-1)*nzpml+(iz-1)]);

// 		// b_txx[index] = b_txx[index] + direction*(scale*2*(lamda[index]+2*miu[index])*(delta_mp[index])*(tmp_vx_x)/dx + \
// 		// 	2*((lamda[index]+2*miu[index])*(delta_mp[index])-2*miu[index]*delta_ms[index])*(tmp_vz_z)/dz);

// 		// b_tzz[index] = b_tzz[index] + direction*(scale*2*((lamda[index]+2*miu[index])*(delta_mp[index])-2*miu[index]*delta_ms[index])*(tmp_vx_x)/dx + \
// 		// 	2*(lamda[index]+2*miu[index])*(delta_mp[index])*(tmp_vz_z)/dz);

// 		// b_txz[index] = b_txz[index] + direction*(scale*2*miu[index]*(delta_ms[index])*((tmp_vz_x)/(4*dx) + (tmp_vx_z)/(4*dz)));


// 		float tmp_vx_x = 0;
// 		float tmp_vz_z = 0;
// 		float tmp_vx_z = 0;
// 		float tmp_vz_x = 0;
// #pragma unroll 4
// 		for(int i=1;i<=nop;i++)
// 		{
// 			 tmp_vx_x += coeff2[i]*(vx[(ix+i-1)*nzpml+iz]-vx[(ix-i)*nzpml+iz]);
// 			 tmp_vz_z += coeff2[i]*(vz[ix*nzpml+(iz+i-1)]-vz[ix*nzpml+(iz-i)]);	

// 			 tmp_vx_z += coeff2[i]*(vx[ix*nzpml+(iz+i)]-vx[ix*nzpml+(iz-i+1)]);
// 			 tmp_vz_x += coeff2[i]*(vz[(ix+i)*nzpml+iz]-vz[(ix-i+1)*nzpml+iz]);				 		
// 		}

// 		b_txx[index] = b_txx[index] + direction*scale*2*(lamda[index]+2*miu[index])*(delta_mp[index])*(tmp_vx_x)/dx + \
// 			2*((lamda[index]+2*miu[index])*(delta_mp[index])-2*miu[index]*delta_ms[index])*(tmp_vz_z)/dz;

// 		b_tzz[index] = b_tzz[index] + direction*scale*2*((lamda[index]+2*miu[index])*(delta_mp[index])-2*miu[index]*delta_ms[index])*(tmp_vx_x)/dx + \
// 			2*(lamda[index]+2*miu[index])*(delta_mp[index])*(tmp_vz_z)/dz;

// 		b_txz[index] = b_txz[index] + direction*scale*2*miu[index]*(delta_ms[index])*((tmp_vz_x)/dx + (tmp_vx_z)/dz);


// }


// __global__ void add_born_source_adjoint(float *txx,float *tzz,float *txz,float *b_vx, float *b_vz,float *delta_mp,float *delta_ms,\
//     const int nxpml, const int nzpml,const int pml,const float dx,const float dz,const int nop,float *lamda,float *miu,float *rho,float *vp,float *vs,int direction)
// {

// 	const int iz = blockIdx.x*blockDim.x+threadIdx.x;
// 	const int ix = blockIdx.y*blockDim.y+threadIdx.y;
// 	// if(iz>=nzpml-nop||ix>=nxpml-nop||iz<nop||ix<nop)return;
// 	if(iz>=nzpml-pml||ix>=nxpml-pml||iz<pml||ix<pml)return;
//     const int index = ix*nzpml+iz;
// //add virtual source 
// 		float scale =1.0;

// 		float tmp_txx_x = 0;
// 		float tmp_txx_z = 0;
// 		float tmp_tzz_x = 0;
// 		float tmp_tzz_z = 0;
// 		float tmp_txz_x = 0;
// 		float tmp_txz_z = 0;

// // TODO: lamda miu inner partial differential
// 		// tmp_txx_x = (txx[(ix+1)*nzpml+iz]-txx[(ix-1)*nzpml+iz])/(2*dx);
// 		// tmp_txx_z = (txx[ix*nzpml+(iz+1)]-txx[ix*nzpml+(iz-1)])/(2*dz);			
// 		// tmp_tzz_x = (tzz[(ix+1)*nzpml+iz]-tzz[(ix-1)*nzpml+iz])/(2*dx);
// 		// tmp_tzz_z = (tzz[ix*nzpml+(iz+1)]-tzz[ix*nzpml+(iz-1)])/(2*dz);	 

// 		// tmp_txz_x = ((txz[(ix)*nzpml+iz] + txz[(ix)*nzpml+(iz-1)]) - (txz[(ix-1)*nzpml+iz] + txz[(ix-1)*nzpml+(iz-1)]))/(2*dx);
// 		// tmp_txz_z = ((txz[(ix)*nzpml+iz] + txz[(ix-1)*nzpml+iz]) - (txz[(ix)*nzpml+(iz-1)] + txz[(ix-1)*nzpml+(iz-1)]))/(2*dz);

// 		// b_vx[index] = b_vx[index] + direction*(1.0/rho[index])*(2*(lamda[index]+2*miu[index])*(delta_mp[index])*(tmp_txx_x) + \
// 		// 	2*((lamda[index]+2*miu[index])*(delta_mp[index])-2*miu[index]*delta_ms[index])*(tmp_tzz_x) + \
// 		// 	2*miu[index]*delta_ms[index]*tmp_txz_z);

// 		// b_vz[index] = b_vz[index] + direction*(1.0/rho[index])*(2*(lamda[index]+2*miu[index])*(delta_mp[index])*(tmp_tzz_z) + \
// 		// 	2*((lamda[index]+2*miu[index])*(delta_mp[index])-2*miu[index]*delta_ms[index])*(tmp_txx_z) + \
// 		// 	2*miu[index]*delta_ms[index]*tmp_txz_x);


// #pragma unroll 4
// 		for(int i=1;i<=nop;i++)
// 		{
// 			 tmp_txx_x += coeff2[i]*(2*(lamda[(ix+i)*nzpml+iz]+2*miu[(ix+i)*nzpml+iz])*(delta_mp[(ix+i)*nzpml+iz])*txx[(ix+i)*nzpml+iz] - \
// 			 2*(lamda[(ix-i+1)*nzpml+iz]+2*miu[(ix-i+1)*nzpml+iz])*(delta_mp[(ix-i+1)*nzpml+iz])*txx[(ix-i+1)*nzpml+iz]);

// 			 tmp_txx_z += coeff2[i]*(2*((lamda[ix*nzpml+(iz+i)]+2*miu[ix*nzpml+(iz+i)])*(delta_mp[ix*nzpml+(iz+i)])-2*miu[ix*nzpml+(iz+i)]*delta_ms[ix*nzpml+(iz+i)])*txx[ix*nzpml+(iz+i)] - \
// 			 2*((lamda[ix*nzpml+(iz-i+1)]+2*miu[ix*nzpml+(iz-i+1)])*(delta_mp[ix*nzpml+(iz-i+1)])-2*miu[ix*nzpml+(iz-i+1)]*delta_ms[ix*nzpml+(iz-i+1)])*txx[ix*nzpml+(iz-i+1)]);	

// 			 tmp_tzz_x += coeff2[i]*(2*((lamda[(ix+i)*nzpml+iz]+2*miu[(ix+i)*nzpml+iz])*(delta_mp[(ix+i)*nzpml+iz])-2*miu[(ix+i)*nzpml+iz]*delta_ms[(ix+i)*nzpml+iz])*tzz[(ix+i)*nzpml+iz] - \
// 			 2*((lamda[(ix-i+1)*nzpml+iz]+2*miu[(ix-i+1)*nzpml+iz])*(delta_mp[(ix-i+1)*nzpml+iz])-2*miu[(ix-i+1)*nzpml+iz]*delta_ms[(ix-i+1)*nzpml+iz])*tzz[(ix-i+1)*nzpml+iz]);

// 			 tmp_tzz_z += coeff2[i]*(2*(lamda[ix*nzpml+(iz+i)]+2*miu[ix*nzpml+(iz+i)])*(delta_mp[ix*nzpml+(iz+i)])*tzz[ix*nzpml+(iz+i)] - \
// 			 2*(lamda[ix*nzpml+(iz-i+1)]+2*miu[ix*nzpml+(iz-i+1)])*(delta_mp[ix*nzpml+(iz-i+1)])*tzz[ix*nzpml+(iz-i+1)]);	

// 			 tmp_txz_x += coeff2[i]*(2*miu[(ix+i-1)*nzpml+iz]*delta_ms[(ix+i-1)*nzpml+iz]*txz[(ix+i-1)*nzpml+iz] - \
// 			 2*miu[(ix+i-1)*nzpml+iz]*delta_ms[(ix+i-1)*nzpml+iz]*txz[(ix-i)*nzpml+iz]);

// 			 tmp_txz_z += coeff2[i]*(2*miu[ix*nzpml+(iz+i-1)]*delta_ms[ix*nzpml+(iz+i-1)]*txz[ix*nzpml+(iz+i-1)] - \
// 			 2*miu[ix*nzpml+(iz-i)]*delta_ms[ix*nzpml+(iz-i)]*txz[ix*nzpml+(iz-i)]);			 		
// 		}
// 		b_vx[index] = b_vx[index] + direction*(1.0/rho[index])*(tmp_txx_x/dx + tmp_tzz_x/dx + tmp_txz_z/dz);

// 		b_vz[index] = b_vz[index] + direction*(1.0/rho[index])*(tmp_tzz_z/dz + tmp_txx_z/dz + tmp_txz_x/dx);

// }



__global__ void add_born_source(float *vx,float *vz,float *b_txx, float *b_tzz,float *b_txz,float *delta_mp,float *delta_ms,\
    const int nxpml, const int nzpml,const int pml,const float dx,const float dz,const int nop,float *lamda,float *miu,float *rho,float *vp,float *vs,int direction)
{

	const int iz = blockIdx.x*blockDim.x+threadIdx.x;
	const int ix = blockIdx.y*blockDim.y+threadIdx.y;
	// if(iz>=nzpml-nop||ix>=nxpml-nop||iz<nop||ix<nop)return;
	if(iz>=nzpml-pml||ix>=nxpml-pml||iz<pml||ix<pml)return;
    const int index = ix*nzpml+iz;
//add virtual source 
		float scale =1.0;

		float tmp_vx_x = 0;
		float tmp_vz_z = 0;
		float tmp_vx_z = 0;
		float tmp_vz_x = 0;
#pragma unroll 4
		for(int i=1;i<=nop;i++)
		{
			 tmp_vx_x += coeff2[i]*(vx[(ix+i-1)*nzpml+iz]-vx[(ix-i)*nzpml+iz]);
			 tmp_vz_z += coeff2[i]*(vz[ix*nzpml+(iz+i-1)]-vz[ix*nzpml+(iz-i)]);	

			 tmp_vx_z += coeff2[i]*(vx[ix*nzpml+(iz+i)]-vx[ix*nzpml+(iz-i+1)]);
			 tmp_vz_x += coeff2[i]*(vz[(ix+i)*nzpml+iz]-vz[(ix-i+1)*nzpml+iz]);				 		
		}

		b_txx[index] = b_txx[index] + direction*scale*2*(lamda[index]+2*miu[index])*(delta_mp[index])*(tmp_vx_x)/dx + \
			2*((lamda[index]+2*miu[index])*(delta_mp[index])-2*miu[index]*(delta_ms[index]))*(tmp_vz_z)/dz;

		b_tzz[index] = b_tzz[index] + direction*scale*2*((lamda[index]+2*miu[index])*(delta_mp[index])-2*miu[index]*(delta_ms[index]))*(tmp_vx_x)/dx + \
			2*(lamda[index]+2*miu[index])*(delta_mp[index])*(tmp_vz_z)/dz;

		b_txz[index] = b_txz[index] + direction*scale*2*miu[index]*(delta_ms[index])*((tmp_vz_x)/dx + (tmp_vx_z)/dz);


}


__global__ void add_born_source_adjoint(float *txx,float *tzz,float *txz,float *b_vx, float *b_vz,float *delta_mp,float *delta_ms,\
    const int nxpml, const int nzpml,const int pml,const float dx,const float dz,const int nop,float *lamda,float *miu,float *rho,float *vp,float *vs,int direction)
{

	const int iz = blockIdx.x*blockDim.x+threadIdx.x;
	const int ix = blockIdx.y*blockDim.y+threadIdx.y;
	// if(iz>=nzpml-nop||ix>=nxpml-nop||iz<nop||ix<nop)return;
	if(iz>=nzpml-pml||ix>=nxpml-pml||iz<pml||ix<pml)return;
    const int index = ix*nzpml+iz;
//add virtual source 
		float scale =1.0;

		float tmp_txx_x = 0;
		float tmp_txx_z = 0;
		float tmp_tzz_x = 0;
		float tmp_tzz_z = 0;
		float tmp_txz_x = 0;
		float tmp_txz_z = 0;

#pragma unroll 4
		for(int i=1;i<=nop;i++)
		{
			 tmp_txx_x += coeff2[i]*(2*(lamda[(ix+i)*nzpml+iz]+2*miu[(ix+i)*nzpml+iz])*(delta_mp[(ix+i)*nzpml+iz])*txx[(ix+i)*nzpml+iz] - \
			 2*(lamda[(ix-i+1)*nzpml+iz]+2*miu[(ix-i+1)*nzpml+iz])*(delta_mp[(ix-i+1)*nzpml+iz])*txx[(ix-i+1)*nzpml+iz]);

			 tmp_txx_z += coeff2[i]*(2*((lamda[ix*nzpml+(iz+i)]+2*miu[ix*nzpml+(iz+i)])*(delta_mp[ix*nzpml+(iz+i)])-2*miu[ix*nzpml+(iz+i)]*(delta_ms[ix*nzpml+(iz+i)]))*txx[ix*nzpml+(iz+i)] - \
			 2*((lamda[ix*nzpml+(iz-i+1)]+2*miu[ix*nzpml+(iz-i+1)])*(delta_mp[ix*nzpml+(iz-i+1)])-2*miu[ix*nzpml+(iz-i+1)]*(delta_ms[ix*nzpml+(iz-i+1)]))*txx[ix*nzpml+(iz-i+1)]);	

			 tmp_tzz_x += coeff2[i]*(2*((lamda[(ix+i)*nzpml+iz]+2*miu[(ix+i)*nzpml+iz])*(delta_mp[(ix+i)*nzpml+iz])-2*miu[(ix+i)*nzpml+iz]*(delta_ms[(ix+i)*nzpml+iz]))*tzz[(ix+i)*nzpml+iz] - \
			 2*((lamda[(ix-i+1)*nzpml+iz]+2*miu[(ix-i+1)*nzpml+iz])*(delta_mp[(ix-i+1)*nzpml+iz])-2*miu[(ix-i+1)*nzpml+iz]*(delta_ms[(ix-i+1)*nzpml+iz]))*tzz[(ix-i+1)*nzpml+iz]);

			 tmp_tzz_z += coeff2[i]*(2*(lamda[ix*nzpml+(iz+i)]+2*miu[ix*nzpml+(iz+i)])*(delta_mp[ix*nzpml+(iz+i)])*tzz[ix*nzpml+(iz+i)] - \
			 2*(lamda[ix*nzpml+(iz-i+1)]+2*miu[ix*nzpml+(iz-i+1)])*(delta_mp[ix*nzpml+(iz-i+1)])*tzz[ix*nzpml+(iz-i+1)]);	

			 tmp_txz_x += coeff2[i]*(2*miu[(ix+i-1)*nzpml+iz]*(delta_ms[(ix+i-1)*nzpml+iz])*txz[(ix+i-1)*nzpml+iz] - \
			 2*miu[(ix+i-1)*nzpml+iz]*(delta_ms[(ix+i-1)*nzpml+iz])*txz[(ix-i)*nzpml+iz]);

			 tmp_txz_z += coeff2[i]*(2*miu[ix*nzpml+(iz+i-1)]*(delta_ms[ix*nzpml+(iz+i-1)])*txz[ix*nzpml+(iz+i-1)] - \
			 2*miu[ix*nzpml+(iz-i)]*(delta_ms[ix*nzpml+(iz-i)])*txz[ix*nzpml+(iz-i)]);			 		
		}
		b_vx[index] = b_vx[index] + direction*(1.0/rho[index])*(tmp_txx_x/dx + tmp_tzz_x/dx + tmp_txz_z/dz);

		b_vz[index] = b_vz[index] + direction*(1.0/rho[index])*(tmp_tzz_z/dz + tmp_txx_z/dz + tmp_txz_x/dx);


}







__global__ void add_born_source_components(float *vx,float *vz,float *b_txx_x,float *b_txx_z, float *b_tzz_x,float *b_tzz_z,float *b_txz_x,float *b_txz_z,float *delta_mp,float *delta_ms,\
    const int nxpml, const int nzpml,const int pml,const float dx,const float dz,const float dt,const int nop,float *lamda,float *miu,float *rho,float *vp,float *vs,int direction)
{

	const int iz = blockIdx.x*blockDim.x+threadIdx.x;
	const int ix = blockIdx.y*blockDim.y+threadIdx.y;
	// if(iz>=nzpml-nop||ix>=nxpml-nop||iz<nop||ix<nop)return;
	if(iz>=nzpml-pml||ix>=nxpml-pml||iz<pml||ix<pml)return;
    const int index = ix*nzpml+iz;
//add virtual source 

		float tmp_vx_x = 0;
		float tmp_vz_z = 0;
		float tmp_vx_z = 0;
		float tmp_vz_x = 0;
#pragma unroll 4
		for(int i=1;i<=nop;i++)
		{
			 tmp_vx_x += coeff2[i]*(vx[(ix+i-1)*nzpml+iz]-vx[(ix-i)*nzpml+iz]);
			 tmp_vz_z += coeff2[i]*(vz[ix*nzpml+(iz+i-1)]-vz[ix*nzpml+(iz-i)]);	

			 tmp_vx_z += coeff2[i]*(vx[ix*nzpml+(iz+i)]-vx[ix*nzpml+(iz-i+1)]);
			 tmp_vz_x += coeff2[i]*(vz[(ix+i)*nzpml+iz]-vz[(ix-i+1)*nzpml+iz]);				 		
		}
//use refl vp and vs
		// b_txx_x[index] = b_txx_x[index] + direction*dt* ( 2*(lamda[index]+2*miu[index])*(delta_mp[index])*(tmp_vx_x)/dx );

		// b_txx_z[index] = b_txx_z[index] + direction*dt* ( 2*((lamda[index]+2*miu[index])*(delta_mp[index])-2*miu[index]*(delta_ms[index]))*(tmp_vz_z)/dz );

		// b_tzz_x[index] = b_tzz_x[index] + direction*dt* ( 2*((lamda[index]+2*miu[index])*(delta_mp[index])-2*miu[index]*(delta_ms[index]))*(tmp_vx_x)/dx );

		// b_tzz_z[index] = b_tzz_z[index] + direction*dt* ( 2*(lamda[index]+2*miu[index])*(delta_mp[index])*(tmp_vz_z)/dz );

		// b_txz_x[index] = b_txz_x[index] + direction*dt* ( 2*miu[index]*(delta_ms[index])*((tmp_vz_x)/dx ) );

		// b_txz_z[index] = b_txz_z[index] + direction*dt* ( 2*miu[index]*(delta_ms[index])*((tmp_vx_z)/dz) );		

// use delta vp and vs
		b_txx_x[index] = b_txx_x[index] + direction*dt* ( 2*(rho[index]*vp[index])*(delta_mp[index])*(tmp_vx_x)/dx );

		b_txx_z[index] = b_txx_z[index] + direction*dt* ((2*(rho[index]*vp[index])*(delta_mp[index]) - 4*(rho[index]*vs[index])*(delta_ms[index]))*(tmp_vz_z)/dz );

		b_tzz_x[index] = b_tzz_x[index] + direction*dt* ( (2*(rho[index]*vp[index])*(delta_mp[index]) - 4*(rho[index]*vs[index])*(delta_ms[index]))*(tmp_vx_x)/dx );

		b_tzz_z[index] = b_tzz_z[index] + direction*dt* ( 2*(rho[index]*vp[index])*(delta_mp[index])*(tmp_vz_z)/dz );

		b_txz_x[index] = b_txz_x[index] + direction*dt* ( ( 2*(rho[index]*vs[index])*(delta_ms[index]))*((tmp_vz_x)/dx ) );

		b_txz_z[index] = b_txz_z[index] + direction*dt* ( ( 2*(rho[index]*vs[index])*(delta_ms[index]))*((tmp_vx_z)/dz) );		



}



__global__ void add_born_source_adjoint_components(float *txx,float *tzz,float *txz,float *b_vx, float *b_vz,float *b_vx_x, float *b_vx_z,float *b_vz_x, float *b_vz_z,float *delta_mp,float *delta_ms,\
    const int nxpml, const int nzpml,const int pml,const float dx,const float dz,const float dt,const int nop,float *lamda,float *miu,float *rho,float *vp,float *vs,int direction)
{

	const int iz = blockIdx.x*blockDim.x+threadIdx.x;
	const int ix = blockIdx.y*blockDim.y+threadIdx.y;
	// if(iz>=nzpml-nop||ix>=nxpml-nop||iz<nop||ix<nop)return;
	if(iz>=nzpml-pml||ix>=nxpml-pml||iz<pml||ix<pml)return;
    const int index = ix*nzpml+iz;
//add virtual source 
		float scale =1.0;

		float tmp_txx_x = 0;
		float tmp_txx_z = 0;
		float tmp_tzz_x = 0;
		float tmp_tzz_z = 0;
		float tmp_txz_x = 0;
		float tmp_txz_z = 0;

#pragma unroll 4
		for(int i=1;i<=nop;i++)
		{

// use delta_vp delta_vs as model parameters
			 tmp_txx_x += coeff2[i]*(2*(rho[(ix+i)*nzpml+iz]*vp[(ix+i)*nzpml+iz])*(delta_mp[(ix+i)*nzpml+iz])*txx[(ix+i)*nzpml+iz] - \
			 2*(rho[(ix-i+1)*nzpml+iz]*vp[(ix-i+1)*nzpml+iz])*(delta_mp[(ix-i+1)*nzpml+iz])*txx[(ix-i+1)*nzpml+iz]);

			 tmp_txx_z += coeff2[i]*(2*((rho[ix*nzpml+(iz+i)]*vp[ix*nzpml+(iz+i)])*(delta_mp[ix*nzpml+(iz+i)])-4*rho[ix*nzpml+(iz+i)]*vs[ix*nzpml+(iz+i)]*(delta_ms[ix*nzpml+(iz+i)]))*txx[ix*nzpml+(iz+i)] - \
			 2*((rho[ix*nzpml+(iz-i+1)]*vp[ix*nzpml+(iz-i+1)])*(delta_mp[ix*nzpml+(iz-i+1)])-4*rho[ix*nzpml+(iz-i+1)]*vs[ix*nzpml+(iz-i+1)]*(delta_ms[ix*nzpml+(iz-i+1)]))*txx[ix*nzpml+(iz-i+1)]);	

			 tmp_tzz_x += coeff2[i]*(2*((rho[(ix+i)*nzpml+iz]*vp[(ix+i)*nzpml+iz])*(delta_mp[(ix+i)*nzpml+iz])-4*rho[(ix+i)*nzpml+iz]*vs[(ix+i)*nzpml+iz]*(delta_ms[(ix+i)*nzpml+iz]))*tzz[(ix+i)*nzpml+iz] - \
			 2*((rho[(ix-i+1)*nzpml+iz]*vp[(ix-i+1)*nzpml+iz])*(delta_mp[(ix-i+1)*nzpml+iz])-4*rho[(ix-i+1)*nzpml+iz]*vs[(ix-i+1)*nzpml+iz]*(delta_ms[(ix-i+1)*nzpml+iz]))*tzz[(ix-i+1)*nzpml+iz]);

			 tmp_tzz_z += coeff2[i]*(2*(rho[ix*nzpml+(iz+i)]*vp[ix*nzpml+(iz+i)])*(delta_mp[ix*nzpml+(iz+i)])*tzz[ix*nzpml+(iz+i)] - \
			 2*(rho[ix*nzpml+(iz-i+1)]*vp[ix*nzpml+(iz-i+1)])*(delta_mp[ix*nzpml+(iz-i+1)])*tzz[ix*nzpml+(iz-i+1)]);	

			 tmp_txz_x += coeff2[i]*(4*rho[(ix+i-1)*nzpml+iz]*vs[(ix+i-1)*nzpml+iz]*(delta_ms[(ix+i-1)*nzpml+iz])*txz[(ix+i-1)*nzpml+iz] - \
			 4*rho[(ix-i)*nzpml+iz]*vs[(ix-i)*nzpml+iz]*(delta_ms[(ix-i)*nzpml+iz])*txz[(ix-i)*nzpml+iz]);

			 tmp_txz_z += coeff2[i]*(4*rho[ix*nzpml+(iz+i-1)]*vs[ix*nzpml+(iz+i-1)]*(delta_ms[ix*nzpml+(iz+i-1)])*txz[ix*nzpml+(iz+i-1)] - \
			 4*rho[ix*nzpml+(iz-i)]*vs[ix*nzpml+(iz-i)]*(delta_ms[ix*nzpml+(iz-i)])*txz[ix*nzpml+(iz-i)]);		

// use refl vp and vs 

			//  tmp_txx_x += coeff2[i]*(2*(lamda[(ix+i)*nzpml+iz]+2*miu[(ix+i)*nzpml+iz])*(delta_mp[(ix+i)*nzpml+iz])*txx[(ix+i)*nzpml+iz] - \
			//  2*(lamda[(ix-i+1)*nzpml+iz]+2*miu[(ix-i+1)*nzpml+iz])*(delta_mp[(ix-i+1)*nzpml+iz])*txx[(ix-i+1)*nzpml+iz]);

			//  tmp_txx_z += coeff2[i]*(2*((lamda[ix*nzpml+(iz+i)]+2*miu[ix*nzpml+(iz+i)])*(delta_mp[ix*nzpml+(iz+i)])-2*miu[ix*nzpml+(iz+i)]*(delta_ms[ix*nzpml+(iz+i)]))*txx[ix*nzpml+(iz+i)] - \
			//  2*((lamda[ix*nzpml+(iz-i+1)]+2*miu[ix*nzpml+(iz-i+1)])*(delta_mp[ix*nzpml+(iz-i+1)])-2*miu[ix*nzpml+(iz-i+1)]*(delta_ms[ix*nzpml+(iz-i+1)]))*txx[ix*nzpml+(iz-i+1)]);	

			//  tmp_tzz_x += coeff2[i]*(2*((lamda[(ix+i)*nzpml+iz]+2*miu[(ix+i)*nzpml+iz])*(delta_mp[(ix+i)*nzpml+iz])-2*miu[(ix+i)*nzpml+iz]*(delta_ms[(ix+i)*nzpml+iz]))*tzz[(ix+i)*nzpml+iz] - \
			//  2*((lamda[(ix-i+1)*nzpml+iz]+2*miu[(ix-i+1)*nzpml+iz])*(delta_mp[(ix-i+1)*nzpml+iz])-2*miu[(ix-i+1)*nzpml+iz]*(delta_ms[(ix-i+1)*nzpml+iz]))*tzz[(ix-i+1)*nzpml+iz]);

			//  tmp_tzz_z += coeff2[i]*(2*(lamda[ix*nzpml+(iz+i)]+2*miu[ix*nzpml+(iz+i)])*(delta_mp[ix*nzpml+(iz+i)])*tzz[ix*nzpml+(iz+i)] - \
			//  2*(lamda[ix*nzpml+(iz-i+1)]+2*miu[ix*nzpml+(iz-i+1)])*(delta_mp[ix*nzpml+(iz-i+1)])*tzz[ix*nzpml+(iz-i+1)]);	

			//  tmp_txz_x += coeff2[i]*(2*miu[(ix+i-1)*nzpml+iz]*(delta_ms[(ix+i-1)*nzpml+iz])*txz[(ix+i-1)*nzpml+iz] - \
			//  2*miu[(ix+i-1)*nzpml+iz]*(delta_ms[(ix+i-1)*nzpml+iz])*txz[(ix-i)*nzpml+iz]);

			//  tmp_txz_z += coeff2[i]*(2*miu[ix*nzpml+(iz+i-1)]*(delta_ms[ix*nzpml+(iz+i-1)])*txz[ix*nzpml+(iz+i-1)] - \
			//  2*miu[ix*nzpml+(iz-i)]*(delta_ms[ix*nzpml+(iz-i)])*txz[ix*nzpml+(iz-i)]);		


		}

		b_vx_x[index] = b_vx_x[index] + direction*dt*((1.0/rho[index])*(tmp_txx_x/dx + tmp_tzz_x/dx ));
		b_vx_z[index] = b_vx_z[index] + direction*dt*((1.0/rho[index])*( tmp_txz_z/dz ));		

		b_vz_x[index] = b_vz_x[index] + direction*dt*((1.0/rho[index])*( tmp_txz_x/dx));
		b_vz_z[index] = b_vz_z[index] + direction*dt*((1.0/rho[index])*(tmp_tzz_z/dz + tmp_txx_z/dz ));		


}





